#include "cudaModelMatrix.h"
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

namespace Markov::API::CUDA{
    __host__ void Markov::API::CUDA::CUDAModelMatrix::MigrateMatrix(){
        hipError_t cudastatus;
        cudastatus = hipMalloc((char ***)&this->device_edgeMatrix, this->matrixSize*sizeof(char*));
        CudaCheckNotifyErr(cudastatus, "Cuda failed to initialize edge matrix.\n");
        for(int i=0;i<this->matrixSize;i++){
            cudastatus = hipMalloc((char **)&this->device_edgeMatrix[i],this->matrixSize*sizeof(char));
            CudaCheckNotifyErr(cudastatus, "Cuda failed to initialize edge matrix row.\n");
        }

        cudastatus = hipMalloc((long int ***)&this->device_valueMatrix, this->matrixSize*sizeof(long int*));
        CudaCheckNotifyErr(cudastatus, "Cuda failed to initialize value matrix row.\n");
        for(int i=0;i<this->matrixSize;i++){
            cudastatus = hipMalloc((char **)&this->device_valueMatrix[i],this->matrixSize*sizeof(long int));
            CudaCheckNotifyErr(cudastatus, "Cuda failed to initialize value matrix row.\n");
        }

        cudastatus = hipMalloc((char**)&this->device_matrixIndex, this->matrixSize*sizeof(char));
        CudaCheckNotifyErr(cudastatus, "Cuda failed to initialize device_matrixIndex.\n");

        cudastatus = hipMalloc((long int **)&this->device_totalEdgeWeights, this->matrixSize*sizeof(long int));
        CudaCheckNotifyErr(cudastatus, "Cuda failed to initialize device_totalEdgeWeights.\n");

        for(int i=0;i<this->matrixSize;i++){
            cudastatus = hipMemcpy(this->device_edgeMatrix[i], this->edgeMatrix[i], this->matrixSize*sizeof(char), hipMemcpyHostToDevice);
            CudaCheckNotifyErr(cudastatus, "Cuda failed to copy to device memory. (edge matrix)\n");
        }

        for(int i=0;i<this->matrixSize;i++){
            cudastatus = hipMemcpy(this->device_valueMatrix[i], this->valueMatrix[i], this->matrixSize*sizeof(long int), hipMemcpyHostToDevice);
            CudaCheckNotifyErr(cudastatus, "Cuda failed to copy to device memory. (value matrix)\n");
        }

        cudastatus = hipMemcpy(this->device_matrixIndex, this->matrixIndex, this->matrixSize*sizeof(char), hipMemcpyHostToDevice);
        CudaCheckNotifyErr(cudastatus, "Cuda failed to copy to device memory. (Index)\n");

        cudastatus = hipMemcpy(this->device_totalEdgeWeights, this->totalEdgeWeights, this->matrixSize*sizeof(long int), hipMemcpyHostToDevice);
        CudaCheckNotifyErr(cudastatus, "Cuda failed to copy to device memory. (Total Edge Values)\n");
        

    }
    __host__ void Markov::API::CUDA::CUDAModelMatrix::RetrieveCudaBuffer(){}
    __host__ void Markov::API::CUDA::CUDAModelMatrix::FastRandomWalk(unsigned long int n, const char* wordlistFileName, int minLen, int maxLen, int threads, bool bFileIO){

    }
}
#include "hip/hip_runtime.h"
#include "cudaModelMatrix.h"
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

namespace Markov::API::CUDA{
    __host__ void Markov::API::CUDA::CUDAModelMatrix::MigrateMatrix(){
        hipError_t cudastatus;
        cudastatus = hipMalloc((char ***)&this->device_edgeMatrix, this->matrixSize*sizeof(char*));
        CudaCheckNotifyErr(cudastatus, "Cuda failed to initialize edge matrix.\n");
        for(int i=0;i<this->matrixSize;i++){
            cudastatus = hipMalloc((char **)&this->device_edgeMatrix[i],this->matrixSize*sizeof(char));
            CudaCheckNotifyErr(cudastatus, "Cuda failed to initialize edge matrix row.\n");
        }

        cudastatus = hipMalloc((long int ***)&this->device_valueMatrix, this->matrixSize*sizeof(long int*));
        CudaCheckNotifyErr(cudastatus, "Cuda failed to initialize value matrix row.\n");
        for(int i=0;i<this->matrixSize;i++){
            cudastatus = hipMalloc((char **)&this->device_valueMatrix[i],this->matrixSize*sizeof(long int));
            CudaCheckNotifyErr(cudastatus, "Cuda failed to initialize value matrix row.\n");
        }

        cudastatus = hipMalloc((char**)&this->device_matrixIndex, this->matrixSize*sizeof(char));
        CudaCheckNotifyErr(cudastatus, "Cuda failed to initialize device_matrixIndex.\n");

        cudastatus = hipMalloc((long int **)&this->device_totalEdgeWeights, this->matrixSize*sizeof(long int));
        CudaCheckNotifyErr(cudastatus, "Cuda failed to initialize device_totalEdgeWeights.\n");

        for(int i=0;i<this->matrixSize;i++){
            cudastatus = hipMemcpy(this->device_edgeMatrix[i], this->edgeMatrix[i], this->matrixSize*sizeof(char), hipMemcpyHostToDevice);
            CudaCheckNotifyErr(cudastatus, "Cuda failed to copy to device memory. (edge matrix)\n");
        }

        for(int i=0;i<this->matrixSize;i++){
            cudastatus = hipMemcpy(this->device_valueMatrix[i], this->valueMatrix[i], this->matrixSize*sizeof(long int), hipMemcpyHostToDevice);
            CudaCheckNotifyErr(cudastatus, "Cuda failed to copy to device memory. (value matrix)\n");
        }

        cudastatus = hipMemcpy(this->device_matrixIndex, this->matrixIndex, this->matrixSize*sizeof(char), hipMemcpyHostToDevice);
        CudaCheckNotifyErr(cudastatus, "Cuda failed to copy to device memory. (Index)\n");

        cudastatus = hipMemcpy(this->device_totalEdgeWeights, this->totalEdgeWeights, this->matrixSize*sizeof(long int), hipMemcpyHostToDevice);
        CudaCheckNotifyErr(cudastatus, "Cuda failed to copy to device memory. (Total Edge Values)\n");
        

    }
    __host__ void Markov::API::CUDA::CUDAModelMatrix::RetrieveCudaBuffer(){}
    __host__ void Markov::API::CUDA::CUDAModelMatrix::FastRandomWalk(unsigned long int n, const char* wordlistFileName, int minLen, int maxLen, int threads, bool bFileIO){

    }
    __host__ void Markov::API::CUDA::CUDAModelMatrix::ListCudaDevices() { //list cuda Capable devices on host.
        int nDevices;
        hipGetDeviceCount(&nDevices);
        for (int i = 0; i < nDevices; i++) {
            hipDeviceProp_t prop;
            hipGetDeviceProperties(&prop, i);
            std::cout << "Device Number: " <<  i  << "\n";
            std::cout << "Device name: " << prop.name << "\n";
            std::cout << "Memory Clock Rate (KHz): " << prop.memoryClockRate << "\n";
            std::cout << "Memory Bus Width (bits): " << prop.memoryBusWidth << "\n";
            std::cout << "Peak Memory Bandwidth (GB/s): " << 2.0 * prop.memoryClockRate * (prop.memoryBusWidth / 8) / 1.0e6 << "\n";
            std::cout << "Max Linear Threads: " << prop.maxThreadsDim[0] << "\n";
            
        }
    }

    __host__ int Markov::API::CUDA::CUDAModelMatrix::CudaCheckNotifyErr(hipError_t _status, const char* msg) {
        if (_status != hipSuccess) {
            std::cerr << "\033[1;31m" << _status << ": " << hipGetErrorString(_status) << "-> "  << msg << "\033[0m" << "\n";
            hipDeviceReset();
            exit(1);
        }
        return 0;
    }
    __global__ static void FastRandomWalkPartition(unsigned long int n, int minLen, int maxLen, bool bFileIO, int threads){

    }
}
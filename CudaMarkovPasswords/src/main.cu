#pragma once
#include <iostream>
#include <string>
#include <cstring>
#include <sstream>
#include <chrono>
#include "cudaModelMatrix.h"
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <>

using Markov::API::CUDA::CUDADeviceController;

int main(int argc, char** argv) {



	Markov::API::CUDA::CUDAModelMatrix markovPass;
	std::cerr << "Importing model.\n";
	markovPass.Import("models/finished.mdl");
	std::cerr << "Import done. \n";
	markovPass.ConstructMatrix();
    //markovPass.DumpJSON();
	CUDADeviceController::ListCudaDevices();

	std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();
	std::cerr << "Starting walk. \n";
	markovPass.FastRandomWalk(1310720000,"/media/ignis/Stuff/wordlist.txt",6,12, false);
	//markovPass.FastRandomWalk(500000000,"/media/ignis/Stuff/wordlist2.txt",6,12,25, true);
	std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();

	std::cerr << "Finished in:" << std::chrono::duration_cast<std::chrono::milliseconds> (end - begin).count() << " milliseconds" << std::endl;
	

}


#include "cudaDeviceController.h"
#include <iostream>

namespace Markov::API::CUDA{
    __host__ void Markov::API::CUDA::CUDADeviceController::ListCudaDevices() { //list cuda Capable devices on host.
        int nDevices;
        hipGetDeviceCount(&nDevices);
        for (int i = 0; i < nDevices; i++) {
            hipDeviceProp_t prop;
            hipGetDeviceProperties(&prop, i);
            std::cout << "Device Number: " <<  i  << "\n";
            std::cout << "Device name: " << prop.name << "\n";
            std::cout << "Memory Clock Rate (KHz): " << prop.memoryClockRate << "\n";
            std::cout << "Memory Bus Width (bits): " << prop.memoryBusWidth << "\n";
            std::cout << "Peak Memory Bandwidth (GB/s): " << 2.0 * prop.memoryClockRate * (prop.memoryBusWidth / 8) / 1.0e6 << "\n";
            std::cout << "Max Linear Threads: " << prop.maxThreadsDim[0] << "\n";
            
        }
    }

    __host__ int Markov::API::CUDA::CUDADeviceController::CudaCheckNotifyErr(hipError_t _status, const char* msg, bool bExit) {
        if (_status != hipSuccess) {
            std::cerr << "\033[1;31m" << msg  << " -> " << hipGetErrorString(_status)  << " ("<< _status << ")" << "\033[0m" << "\n";
            
            if(bExit) {
                hipDeviceReset();
                exit(1);
            }
        }
        return 0;
    }

/*
    template <typename T>
    __host__ hipError_t Markov::API::CUDA::CUDADeviceController::CudaMalloc2DToFlat(T* dst, int row, int col){
        return  hipMalloc((T **)&dst, row*col*sizeof(T));
    }

    template <typename T>
    __host__ hipError_t Markov::API::CUDA::CUDADeviceController::CudaMemcpy2DToFlat(T* dst, T** src, int row, int col){
         hipError_t cudastatus;
         for(int i=0;i<row;i++){
            cudastatus = hipMemcpy(dst + (i*col*sizeof(T)), 
                src[i], col*sizeof(T), hipMemcpyHostToDevice);
            if(cudastatus != hipSuccess) return cudastatus;
        }
        return hipSuccess;
    }
*/

};